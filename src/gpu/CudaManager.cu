// CudaManager.cu
#include "CudaManager.hpp"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <memory>
#include <atomic>


namespace
{

std::atomic< unsigned long long > count = 0;

} // namespace



CudaManager::CudaManager( const bool print )
  : print_( print )
{
  if ( count == 0 )
  {
    // use device with highest Gflops/s
    int devID = findCudaDevice( 0, 0, print_ );

    if ( devID < 0 )
    {
      throw std::runtime_error( "No CUDA capable devices found" );
    }
  }
  ++count;
}


CudaManager::~CudaManager( )
{
  if ( count == 1 )
  {
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset( );

    if ( print_ )
    {
      std::cout << "CUDA device reset" << std::endl;
    }
  }
  --count;
}

