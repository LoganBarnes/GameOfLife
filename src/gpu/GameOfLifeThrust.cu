// GameOfLifeThrust.cu
#include "GameOfLifeThrust.hpp"

#include "GameOfLifeAlgorithm.hpp"
#include "CudaManager.hpp"

#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>

#include <stdexcept>


namespace gol
{


///
/// \brief The PropogateFunctor struct
///
struct PropogateFunctor
{
  const GolBool *d_prev;
  const dim3 dim;

  PropogateFunctor(
                   const GolBool *d_prev_,
                   const dim3     dim_
                   )
    : d_prev( d_prev_ )
    , dim   ( dim_ )
  {}

  ///
  /// \brief operator ()
  /// \param t
  ///
  template< typename Tuple >
  __device__
  void
  operator()( Tuple t ) const
  {
    // get neighbors
    uint idx = thrust::get< 0 >( t );
    uint x   = idx % dim.x;
    uint y   = idx / dim.x;

    thrust::get< 1 >( t ) = findNeighbors( d_prev, dim, x, y );
  }


};



///
/// \brief The GameOfLifeThrust::GoLThrustImpl class
///
class GameOfLifeThrust::GoLThrustImpl
{
public:

  explicit
  GoLThrustImpl(
                const std::vector< GolBool >      &initState,
                std::vector< GolBool >::size_type width,
                std::vector< GolBool >::size_type height
                );

  ~GoLThrustImpl( ) = default;

  void propogateState ( );

  const thrust::device_vector< GolBool > &getState ( );

  bool
  updateSinceGetState( ) const { return updateSinceGetState_; }


private:

  thrust::device_vector< GolBool > dCurrState_;
  thrust::device_vector< GolBool > dPrevState_;

  std::vector< float >::size_type width_, height_;

  bool updateSinceGetState_;

};



///
/// \brief GameOfLifeThrust::GoLThrustImpl::GoLThrustImpl
/// \param initState
/// \param width
///
GameOfLifeThrust::GoLThrustImpl::GoLThrustImpl(
                                               const std::vector< GolBool >      &initState,
                                               std::vector< GolBool >::size_type width,
                                               std::vector< GolBool >::size_type height
                                               )
  : dCurrState_( initState.size( ) )
  , dPrevState_( dCurrState_.size( ) )
  , width_( width )
  , height_( height )
  , updateSinceGetState_( true )
{
  checkCudaErrors( hipMemcpy(
                              thrust::raw_pointer_cast( dCurrState_.data( ) ),
                              initState.data( ),
                              dCurrState_.size( ) * sizeof( GolBool ),
                              hipMemcpyHostToDevice
                              ) );
}



///
/// \brief GameOfLifeThrust::GoLThrustImpl::propogateState
///
void
GameOfLifeThrust::GoLThrustImpl::propogateState( )
{
  dPrevState_.swap( dCurrState_ ); // O(1) just swaps pointers

  dim3 dim( static_cast< unsigned >( width_ ), static_cast< unsigned >( height_ ) );

  thrust::counting_iterator< SizeType > first( 0 );
  thrust::counting_iterator< SizeType > last( dCurrState_.size( ) );

  thrust::for_each(
                   thrust::make_zip_iterator( thrust::make_tuple( first, dCurrState_.begin( ) ) ),
                   thrust::make_zip_iterator( thrust::make_tuple( last,  dCurrState_.end( ) ) ),
                   PropogateFunctor( thrust::raw_pointer_cast( dPrevState_.data( ) ), dim )
                   );

  updateSinceGetState_ = true;
} // propogateState



///
/// \brief GameOfLifeThrust::GoLThrustImpl::getState
/// \return
///
const thrust::device_vector< GolBool >&
GameOfLifeThrust::GoLThrustImpl::getState( )
{
  updateSinceGetState_ = false;
  return dCurrState_;
}



///
/// \brief GameOfLifeThrust::GameOfLifeThrust
/// \param initState
/// \param width
///
GameOfLifeThrust::GameOfLifeThrust(
                                   std::vector< GolBool >            initState,
                                   std::vector< GolBool >::size_type width,
                                   std::vector< GolBool >::size_type height
                                   )
  : GameOfLife( initState, width, height )
  , cuda_( )
  , upImpl_( new GameOfLifeThrust::GoLThrustImpl(
                                                 state_,
                                                 width,
                                                 height
                                                 ) )
{}



///
/// \brief GameOfLifeThrust::~GameOfLifeThrust
///
GameOfLifeThrust::~GameOfLifeThrust( )
{}



///
/// \brief GameOfLifeThrust::propogateState
///
void
GameOfLifeThrust::propogateState( )
{
  upImpl_->propogateState( );
}



///
/// \brief GameOfLifeThrust::getState
/// \return
///
const std::vector< GolBool >&
GameOfLifeThrust::getState( )
{
  const bool updateSinceGetState = upImpl_->updateSinceGetState( );

  const thrust::device_vector< GolBool > &dState = upImpl_->getState( );

  if ( updateSinceGetState )
  {
    hipDeviceSynchronize( );
    thrust::copy( dState.begin( ), dState.end( ), state_.begin( ) );
  }

  return state_;
}



} // namespace gol
