// GameOfLifeThrust.cu
#include "GameOfLifeThrust.hpp"

#include <hip/hip_runtime_api.h>
#include <helper_grid.h>

// this prevents nvcc from causing warnings
// in thirdparty headers on windows
#pragma warning(push, 0)
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <stdexcept>
#pragma warning(pop)


namespace gol
{


namespace
{

///
/// \brief The CudaPrep struct
///
struct CudaPrep
{

  CudaPrep( )
  {
    // use device with highest Gflops/s
    int devID = findCudaDevice( 0, 0, false );

    if ( devID < 0 )
    {
      throw std::runtime_error( "No CUDA capable devices found" );
    }

    std::cout << "CUDA device initialized" << std::endl;
  }


  ~CudaPrep( )
  {
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset( );
    std::cout << "CUDA device reset" << std::endl;
  }


};

///
/// \brief cudaPrep
///
const CudaPrep cudaPrep;

} // namespace



///
/// \brief propogateState_k
/// \param pPrev
/// \param dim
/// \param x
/// \param y
/// \return
///
__device__
char
propogateState_k(
                 const char *pPrev,
                 const dim3  dim,
                 const uint  x,
                 const uint  y
                 )
{
  uint neighbors = 0;

  // find number of living neighbors
  // top row
  uint iy = ( y + dim.y - 1 ) % dim.y;
  uint ix = ( x + dim.x - 1 ) % dim.x;

  neighbors += ( pPrev[ iy * dim.x + ix ] ? 1 : 0 );

  ix         = x;
  neighbors += ( pPrev[ iy * dim.x + ix ] ? 1 : 0 );

  ix         = ( x + 1 ) % dim.x;
  neighbors += ( pPrev[ iy * dim.x + ix ] ? 1 : 0 );

  // middle row
  iy         = y;
  ix         = ( x + dim.x - 1 ) % dim.x;
  neighbors += ( pPrev[ iy * dim.x + ix ] ? 1 : 0 );

  ix         = ( x + 1 ) % dim.x;
  neighbors += ( pPrev[ iy * dim.x + ix ] ? 1 : 0 );

  // bottom row
  iy         = ( y + 1 ) % dim.y;
  ix         = ( x + dim.x - 1 ) % dim.x;
  neighbors += ( pPrev[ iy * dim.x + ix ] ? 1 : 0 );

  ix         = x;
  neighbors += ( pPrev[ iy * dim.x + ix ] ? 1 : 0 );

  ix         = ( x + 1 ) % dim.x;
  neighbors += ( pPrev[ iy * dim.x + ix ] ? 1 : 0 );

  char state = pPrev[ y * dim.x + x ];

  if ( state && ( neighbors != 2 && neighbors != 3 ) )
  {
    return false;
  }
  else
  if ( !state && neighbors == 3 )
  {
    return true;
  }

  return state;
} // propogateState_k



///
/// \brief The PropogateFunctor struct
///
struct PropogateFunctor
{
  const char *d_prev;
  const dim3 dim;

  PropogateFunctor(
                   const char *d_prev_,
                   const dim3  dim_
                   )
    : d_prev( d_prev_ )
    , dim( dim_ )
  {}

  ///
  /// \brief operator ()
  /// \param t
  ///
  template< typename Tuple >
  __device__
  void
  operator()( Tuple t ) const
  {
    // get neighbors
    uint idx = thrust::get< 0 >( t );
    uint x   = idx % dim.x;
    uint y   = idx / dim.x;

    thrust::get< 1 >( t ) = propogateState_k( d_prev, dim, x, y );
  }


};



///
/// \brief The GameOfLifeThrust::GoLThrustImpl class
///
class GameOfLifeThrust::GoLThrustImpl
{
public:

  explicit
  GoLThrustImpl(
                const std::vector< char >      &initState,
                std::vector< char >::size_type width,
                std::vector< char >::size_type height
                );

  ~GoLThrustImpl( ) = default;

  void propogateState ( );

  const thrust::device_vector< char > &getState ( );

  char updateSinceGetState ( ) const { return updateSinceGetState_; }


private:

  thrust::device_vector< char > dCurrState_;
  thrust::device_vector< char > dPrevState_;

  std::vector< float >::size_type width_, height_;

  char updateSinceGetState_;

};



///
/// \brief GameOfLifeThrust::GoLThrustImpl::GoLThrustImpl
/// \param initState
/// \param width
///
GameOfLifeThrust::GoLThrustImpl::GoLThrustImpl(
                                               const std::vector< char >      &initState,
                                               std::vector< char >::size_type width,
                                               std::vector< char >::size_type height
                                               )
  : dCurrState_( initState.size( ) )
  , dPrevState_( dCurrState_.size( ) )
  , width_( width )
  , height_( height )
  , updateSinceGetState_( true )
{
  checkCudaErrors( hipMemcpy(
                              thrust::raw_pointer_cast( dCurrState_.data() ),
                              initState.data( ),
                              dCurrState_.size() * sizeof( char ),
                              hipMemcpyHostToDevice
                              ) );
}



///
/// \brief GameOfLifeThrust::GoLThrustImpl::propogateState
///
void
GameOfLifeThrust::GoLThrustImpl::propogateState( )
{
  dPrevState_.swap( dCurrState_ ); // O(1) just swaps pointers

  dim3 dim( static_cast< unsigned >( width_), static_cast< unsigned >( height_ ) );

  thrust::counting_iterator< uint > first( 0 );
  thrust::counting_iterator< uint > last = first + dCurrState_.size( );

  thrust::for_each(
                   thrust::make_zip_iterator( thrust::make_tuple( first, dCurrState_.begin( ) ) ),
                   thrust::make_zip_iterator( thrust::make_tuple( last, dCurrState_.end( ) ) ),
                   PropogateFunctor( thrust::raw_pointer_cast( dPrevState_.data( ) ), dim )
                   );

  updateSinceGetState_ = true;
}



///
/// \brief GameOfLifeThrust::GoLThrustImpl::getState
/// \return
///
const thrust::device_vector< char >&
GameOfLifeThrust::GoLThrustImpl::getState( )
{
  updateSinceGetState_ = false;
  return dCurrState_;
}



///
/// \brief GameOfLifeThrust::GameOfLifeThrust
/// \param initState
/// \param width
///
GameOfLifeThrust::GameOfLifeThrust(
                                   std::vector< char >            initState,
                                   std::vector< char >::size_type width,
                                   std::vector< char >::size_type height
                                   )
  : GameOfLife( initState, width, height )
  , upImpl_( new GameOfLifeThrust::GoLThrustImpl(
                                                 state_,
                                                 width,
                                                 height
                                                 ) )
{}



///
/// \brief GameOfLifeThrust::propogateState
///
void
GameOfLifeThrust::propogateState( )
{
  upImpl_->propogateState( );
}



///
/// \brief GameOfLifeThrust::getState
/// \return
///
const std::vector< char >&
GameOfLifeThrust::getState( )
{
  const thrust::device_vector< char > &dState = upImpl_->getState( );

  if ( upImpl_->updateSinceGetState( ) )
  {
    hipDeviceSynchronize( );
    thrust::copy( dState.begin( ), dState.end( ), state_.begin( ) );
  }

  return state_;
}



} // namespace gol
